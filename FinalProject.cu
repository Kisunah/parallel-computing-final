#include "hip/hip_runtime.h"
// Seth Hanusik
// Parallel Computing
// Final Project

#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <fstream>
#include <chrono>
#include <cmath>
#include <thread>

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/sort.h>

using namespace std;

struct Table {
    int stateId;
    int parentStateId;
    int priority;
    int currArr[18];
    char previousMove;
    int totalCost = 0;
    int f;
    int g;
    int h;

    Table();
    Table(int *arr);
};

Table::Table(int *arr) {
    // currArr = arr;
    memcpy(currArr, arr, 18 * sizeof(int));
    priority = 0;
    previousMove = 'X';
    f = 0;
    g = 0;
    h = 0;
}

Table::Table() {
    priority = 0;
    previousMove = 'X';
    f = 0;
    g = 0;
    h = 0;
}

bool checkGoal(Table parentTable, Table goalTable) {
    int* curr = parentTable.currArr;
    int* goal = goalTable.currArr;

    bool equal = true;
    for (int i = 0; i < 18; i++) {
        if (curr[i] != goal[i]) {
            equal = false;
        }
    }

    return equal;
}

int getOpenPosition(Table currTable) {
    int* currVec = currTable.currArr;
    int zeroPosition;

    for (int i = 0; i < 18; i++) {
        if (currVec[i] == 0) {
            zeroPosition = i;
        }
    }

    return zeroPosition;
}

__global__
void generateTables(int openPosition, Table currTable, Table goalTable, int* d_id, Table *childArr, bool *d_left, bool *d_up, bool *d_right, bool *d_down) {
    if (openPosition % 3 != 0 && currTable.previousMove != 'L' && threadIdx.x == 0) {
        int parentCost = currTable.totalCost;
        int movingValue = currTable.currArr[openPosition - 1];
        int *leftArr = (int *)malloc(sizeof(int)*18);
        memcpy(leftArr, currTable.currArr, 18*sizeof(int));
        leftArr[openPosition] = movingValue;
        leftArr[openPosition - 1] = 0;

        Table *leftChild = &childArr[0];
        memcpy(leftChild->currArr, leftArr, 18 * sizeof(int));
        atomicAdd(d_id, 1);
        leftChild->stateId = *d_id;
        leftChild->parentStateId = currTable.stateId;
        leftChild->previousMove = 'R';

        if (movingValue < 7) {
            leftChild->totalCost = parentCost + 1;
        } else if (movingValue < 17) {
            leftChild->totalCost = parentCost + 3;
        } else {
            leftChild->totalCost = parentCost + 15;
        }

        int f = 0;
        int g = leftChild->totalCost;
        int h = 0;

        for (int i = 0; i < 18; i++) {
            if (leftArr[i] != goalTable.currArr[i]) {
                h++;
            }
        }
        f = g + h;
        leftChild->priority = f;

        leftChild->f = f;
        leftChild->g = g;
        leftChild->h = g;

        childArr[0] = *leftChild;
        *d_left = true;
    }

    if (openPosition > 2 && currTable.previousMove != 'U' && threadIdx.x == 1) {
        int parentCost = currTable.totalCost;
        int movingValue = currTable.currArr[openPosition - 3];
        int *upArr = (int *)malloc(sizeof(int)*18);
        memcpy(upArr, currTable.currArr, 18*sizeof(int));
        upArr[openPosition] = movingValue;
        upArr[openPosition - 3] = 0;

        Table *upChild = &childArr[1];
        memcpy(upChild->currArr, upArr, 18* sizeof(18));
        atomicAdd(d_id, 1);
        upChild->stateId = *d_id;
        upChild->parentStateId = currTable.stateId;
        upChild->previousMove = 'U';

        if (movingValue < 7) {
            upChild->totalCost = parentCost + 1;
        } else if (movingValue < 17) {
            upChild->totalCost = parentCost + 3;
        } else {
            upChild->totalCost = parentCost + 15;
        }

        int f = 0;
        int g = upChild->totalCost;
        int h = 0;

        for (int i = 0; i < 18; i++) {
            if (upArr[i] != goalTable.currArr[i]) {
                h++;
            }
        }
        f = g + h;
        upChild->priority = f;

        upChild->f = f;
        upChild->g = g;
        upChild->h = h;

        childArr[1] = *upChild;
        *d_up = true;
    }

    if (openPosition % 3 != 2 && currTable.previousMove != 'R' && threadIdx.x == 2) {
        int parentCost = currTable.totalCost;
        int movingValue = currTable.currArr[openPosition + 1];
        int *rightArr = (int *)malloc(sizeof(int)*18);
        memcpy(rightArr, currTable.currArr, 18*sizeof(int));
        rightArr[openPosition] = movingValue;
        rightArr[openPosition + 1] = 0;

        Table *rightChild = &childArr[2];
        memcpy(rightChild->currArr, rightArr, 18*sizeof(int));
        atomicAdd(d_id, 1);
        rightChild->stateId = *d_id;
        rightChild->parentStateId = currTable.stateId;
        rightChild->previousMove = 'R';

        if (movingValue < 7) {
            rightChild->totalCost = parentCost + 1;
        } else if (movingValue < 17) {
            rightChild->totalCost = parentCost + 3;
        } else {
            rightChild->totalCost = parentCost + 15;
        }

        int f = 0;
        int g = rightChild->totalCost;
        int h = 0;

        for (int i = 0; i < 18; i++) {
            if (rightArr[i] != goalTable.currArr[i]) {
                h++;
            }
        }
        f = g + h;
        rightChild->priority = f;

        rightChild->f = f;
        rightChild->g = g;
        rightChild->h = h;

        childArr[2] = *rightChild;
        *d_right = true;
    }

    if (openPosition < 15 && currTable.previousMove != 'D' && threadIdx.x == 3) {
        int parentCost = currTable.totalCost;
        int movingValue = currTable.currArr[openPosition + 3];
        int *downArr = (int *)malloc(sizeof(int)*18);
        memcpy(downArr, currTable.currArr, 18*sizeof(int));
        downArr[openPosition] = movingValue;
        downArr[openPosition + 3] = 0;


        Table *downChild = &childArr[3];
        memcpy(downChild->currArr, downArr, 18*sizeof(18));
        atomicAdd(d_id, 1);
        downChild->stateId = *d_id;
        downChild->parentStateId = currTable.stateId;
        downChild->previousMove = 'D';

        if (movingValue < 7) {
            downChild->totalCost = parentCost + 1;
        } else if (movingValue < 17) {
            downChild->totalCost = parentCost + 3;
        } else {
            downChild->totalCost = parentCost + 15;
        }

        int f = 0;
        int g = downChild->totalCost;
        int h = 0;

        for (int i = 0; i < 18; i++) {
            if (downArr[i] != goalTable.currArr[i]) {
                h++;
            }
        }
        f = g + h;
        downChild->priority = f;

        downChild->f = f;
        downChild->g = g;
        downChild->h = h;

        childArr[3] = *downChild;
        *d_down = true;
    }

    __syncthreads();
}

void printTable(Table table) {
    int *arr = table.currArr;
    for (int i = 0; i < 18; i++) {
        if (i % 3 == 0 && i != 0) {
            printf("\n");
        }
        printf("%d ", arr[i]);
    }
    printf("\n");
}

thrust::host_vector<Table> sortOpenList(thrust::host_vector<Table> openList) {
    thrust::sort(openList.begin(), openList.end(), [ ](Table lhs, Table rhs) {
        return lhs.priority > rhs.priority;
    });
    return openList;
}

void solve(Table goalTable, int *d_startArr, int *d_goalArr, thrust::host_vector<Table> openList, thrust::host_vector<Table> closedList, int* d_id) {
    bool equal = checkGoal(openList[0], goalTable);
    Table currTable;

    while (!equal) {
        currTable = openList[openList.size() - 1];
        equal = checkGoal(currTable, goalTable);
        if (!equal) {
            openList.pop_back();
            int openPosition = getOpenPosition(currTable);
            int numTablesGenerated = -1;
            int *d_numTablesGenerated;
            hipMalloc((void**)&d_numTablesGenerated, sizeof(int));
            hipMemcpy(d_numTablesGenerated, &numTablesGenerated, sizeof(int), hipMemcpyHostToDevice);
            
            int n = 4;
            Table *childArr, *d_childArr;
            childArr = (Table*)malloc(sizeof(Table)*n);
            for (int i = 0; i < n; i++) {
                int *tempArr = currTable.currArr;
                memcpy(childArr[i].currArr, tempArr, 18 * sizeof(int));

                childArr[i].priority = 0;
                childArr[i].f = 0;
                childArr[i].g = 0;
                childArr[i].h = 0;
            }

            hipMalloc((void**)&d_childArr, n * sizeof(Table));
            hipMemcpy(d_childArr, childArr, n * sizeof(Table), hipMemcpyHostToDevice);

            bool *d_left, *d_up, *d_right, *d_down;
            bool left = false;
            bool up = false;
            bool right = false;
            bool down = false;
            hipMalloc((void**)&d_left, sizeof(bool));
            hipMemcpy(d_left, &left, sizeof(bool), hipMemcpyHostToDevice);
            hipMalloc((void**)&d_up, sizeof(bool));
            hipMemcpy(d_up, &up, sizeof(bool), hipMemcpyHostToDevice);
            hipMalloc((void**)&d_right, sizeof(bool));
            hipMemcpy(d_right, &right, sizeof(bool), hipMemcpyHostToDevice);
            hipMalloc((void**)&d_down, sizeof(bool));
            hipMemcpy(d_down, &d_down, sizeof(bool), hipMemcpyHostToDevice);

            generateTables<<<1,4>>>(openPosition, currTable, goalTable, d_id, d_childArr, d_left, d_up, d_right, d_down);
            hipDeviceSynchronize();
            hipMemcpy(childArr, d_childArr, 4 * sizeof(Table), hipMemcpyDeviceToHost);
            hipMemcpy(&left, d_left, sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(&up, d_up, sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(&right, d_right, sizeof(bool), hipMemcpyDeviceToHost);
            hipMemcpy(&down, d_down, sizeof(bool), hipMemcpyDeviceToHost);

            if (left) {
                openList.push_back(childArr[0]);
            }
            if (up) {
                openList.push_back(childArr[1]);
            }
            if (right) {
                openList.push_back(childArr[2]);
            }
            if (down) {
                openList.push_back(childArr[3]);
            }
            openList = sortOpenList(openList);
            closedList.push_back(currTable);
        }
    }
    closedList.push_back(currTable);
}

int main() {
    thrust::host_vector<int> startVec(18);
    thrust::host_vector<int> goalVec(18);

    startVec[0] = 3;
    startVec[1] = 1;
    startVec[2] = 2;
    startVec[3] = 6;
    startVec[4] = 4;
    startVec[5] = 5;
    startVec[6] = 9;
    startVec[7] = 7;
    startVec[8] = 8;
    startVec[9] = 10;
    startVec[10] = 0;
    startVec[11] = 11;
    startVec[12] = 12;
    startVec[13] = 13;
    startVec[14] = 14;
    startVec[15] = 15;
    startVec[16] = 16;
    startVec[17] = 17;

    goalVec[0] = 0;
    goalVec[1] = 1;
    goalVec[2] = 2;
    goalVec[3] = 3;
    goalVec[4] = 4;
    goalVec[5] = 5;
    goalVec[6] = 6;
    goalVec[7] = 7;
    goalVec[8] = 8;
    goalVec[9] = 9;
    goalVec[10] = 10;
    goalVec[11] = 11;
    goalVec[12] = 12;
    goalVec[13] = 13;
    goalVec[14] = 14;
    goalVec[15] = 15;
    goalVec[16] = 16;
    goalVec[17] = 17;

    thrust::device_vector<int> d_startVec = startVec;
    thrust::device_vector<int> d_goalVec = goalVec;

    int* startArr = thrust::raw_pointer_cast(&startVec[0]);
    int* goalArr = thrust::raw_pointer_cast(&goalVec[0]);
    int* d_startArr = thrust::raw_pointer_cast(&d_startVec[0]);
    int* d_goalArr = thrust::raw_pointer_cast(&d_goalVec[0]);

    Table startTable = Table(startArr);
    startTable.stateId = 1;
    Table goalTable = Table(goalArr);
    Table *d_startTable, *d_goalTable;
    hipMemcpy(d_startTable, &startTable, sizeof(Table), hipMemcpyHostToDevice);
    hipMemcpy(d_goalTable, &goalTable, sizeof(Table), hipMemcpyHostToDevice);

    thrust::host_vector<Table> openList(0);
    thrust::host_vector<Table> closedList(0);
    openList.push_back(startTable);

    int id = 1;
    int *d_id;
    hipMalloc((void**)&d_id, sizeof(int));
    hipMemcpy(d_id, &id, sizeof(int), hipMemcpyHostToDevice);

    auto start = std::chrono::steady_clock::now();

    solve(goalTable, d_startArr, d_goalArr, openList, closedList, d_id);

    auto end = std::chrono::steady_clock::now();
    std::chrono::duration<double> elapsed_seconds = end-start;
    cout << "Time: " << elapsed_seconds.count() << endl;
    return 0;
}